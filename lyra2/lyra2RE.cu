#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint64_t* d_hash[MAX_GPUS];
//static uint64_t* d_hash2[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

//extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *hash);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_setTarget(const void *ptarget);
extern uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern uint32_t groestl256_getSecNonce(int thr_id, int num);

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("lyra %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

extern "C" void lyra2re_hash(void *state, const void *input)
{
	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 18 : 17;
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << intensity); // 18=256*256*4;
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);

		// DMatrix
//		hipMalloc(&d_hash2[thr_id], (size_t)16 * 8 * 8 * sizeof(uint64_t) * throughput);
//		lyra2_cpu_init(thr_id, throughput, d_hash2[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		*hashes_done = pdata[19] - first_nonce + throughput;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE("S")

		foundNonce = groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];

			be32enc(&endiandata[19], foundNonce);
			lyra2re_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = groestl256_getSecNonce(thr_id, 1);
				if (secNonce != UINT32_MAX)
				{
					be32enc(&endiandata[19], secNonce);
					lyra2re_hash(vhash64, endiandata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						if (opt_debug)
							applog(LOG_BLUE, "GPU #%d: found second nonce %08x", device_map[thr_id], secNonce);
						pdata[21] = secNonce;
						res++;
					}
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	return 0;
}
