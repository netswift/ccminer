#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#define TPB52 8
#define TPB50 16

#include "cuda_lyra2v2_sm3.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#if __CUDA_ARCH__ >= 500

#include "cuda_lyra2_vectors.h"

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DMatrix;

__device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d ^= a; d = SWAPUINT2(d);
	c += d; b ^= c; b = ROR2(b, 24);
	a += b; d ^= a; d = ROR2(d, 16);
	c += d; b ^= c; b = ROR2(b, 63);
}

__device__ __forceinline__
void round_lyra_v5(uint2x4* s)
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}

__device__ __forceinline__
void reduceDuplex(uint2x4 state[4], const uint32_t thread)
{
	uint2x4 state1[3];
	const uint32_t ps1 = (Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * (Ncol-1) + memshift * Ncol + Nrow * Ncol * memshift * thread);

	#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 - i*memshift;

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra_v5(state);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];
	}
}

__device__ __forceinline__
void reduceDuplex50(uint2x4 state[4], const uint32_t thread)
{
	const uint32_t ps1 = (Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * (Ncol - 1) + memshift * Ncol + Nrow * Ncol * memshift * thread);

	#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const int32_t s2 = ps2 - i*memshift;

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]);

		round_lyra_v5(state);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = __ldg4(&(DMatrix + s1)[j]) ^ state[j];
	}
}

__device__ __forceinline__
void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, uint2x4 state[4], const uint32_t thread)
{
	uint2x4 state2[3], state1[3];

	const uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
	const uint32_t ps3 = (memshift * (Ncol-1) + memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);

	for (int i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 - i*memshift;

#if __CUDA_ARCH__ == 500

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] = state[j] ^ (__ldg4(&(DMatrix + s1)[j]) + __ldg4(&(DMatrix + s2)[j]));

		round_lyra_v5(state);
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

#else /* 5.2 */

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);
		#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			uint2x4 tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}

		round_lyra_v5(state);

		#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

#endif
		((uint2*)state2)[0] ^= ((uint2*)state)[11];

		#pragma unroll
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];

		#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}


__device__ __forceinline__
void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, uint2x4* state, const uint32_t thread)
{
	uint2x4 state1[3], state2[3];
	const uint32_t ps1 = (memshift * Ncol * rowIn    + Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
	const uint32_t ps3 = (memshift * Ncol * rowOut   + Nrow * Ncol * memshift * thread);

	for (int i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 + i*memshift;

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] += state2[j];

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra_v5(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];

		#pragma unroll
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

#if __CUDA_ARCH__ == 500
		if (rowInOut != rowOut)
		{
			#pragma unroll
			for (int j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		}
		if (rowInOut == rowOut)
		{
			#pragma unroll
			for (int j = 0; j < 3; j++)
				state2[j] ^= state[j];
		}
#else
		if (rowInOut != rowOut)
		{
			#pragma unroll
			for (int j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];
		} else {
			#pragma unroll
			for (int j = 0; j < 3; j++)
				state2[j] ^= state[j];
		}
#endif
		#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}


#if __CUDA_ARCH__ == 500
__global__ __launch_bounds__(TPB50, 1)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void lyra2v2_gpu_hash_32(const uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint2x4 blake2b_IV[2];

	if (threadIdx.x == 0) {

		((uint16*)blake2b_IV)[0] = make_uint16(
			0xf3bcc908, 0x6a09e667, 0x84caa73b, 0xbb67ae85,
			0xfe94f82b, 0x3c6ef372, 0x5f1d36f1, 0xa54ff53a,
			0xade682d1, 0x510e527f, 0x2b3e6c1f, 0x9b05688c,
			0xfb41bd6b, 0x1f83d9ab, 0x137e2179, 0x5be0cd19
		);
	}

	if (thread < threads)
	{
		uint2x4 state[4];

		((uint2*)state)[0] = __ldg(&g_hash[thread]);
		((uint2*)state)[1] = __ldg(&g_hash[thread + threads]);
		((uint2*)state)[2] = __ldg(&g_hash[thread + threads*2]);
		((uint2*)state)[3] = __ldg(&g_hash[thread + threads*3]);

		state[1] = state[0];

		state[2] = ((blake2b_IV)[0]);
		state[3] = ((blake2b_IV)[1]);

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		((uint2*)state)[0].x ^= 0x20;
		((uint2*)state)[1].x ^= 0x20;
		((uint2*)state)[2].x ^= 0x20;
		((uint2*)state)[3].x ^= 0x01;
		((uint2*)state)[4].x ^= 0x04;
		((uint2*)state)[5].x ^= 0x04;
		((uint2*)state)[6].x ^= 0x80;
		((uint2*)state)[7].y ^= 0x01000000;

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		const uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread);

		for (int i = 0; i < Ncol; i++)
		{
			const uint32_t s1 = ps1 - memshift * i;
			DMatrix[s1] = state[0];
			DMatrix[s1+1] = state[1];
			DMatrix[s1+2] = state[2];
			round_lyra_v5(state);
		}

		reduceDuplex50(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state, thread);
		reduceDuplexRowSetupV2(2, 1, 3, state, thread);

		uint32_t rowa;
		int prev=3;

		for (int i = 0; i < 4; i++)
		{
			rowa = ((uint2*)state)[0].x & 3;
			reduceDuplexRowtV2(prev, rowa, i, state, thread);
			prev = i;
		}

		const uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
			round_lyra_v5(state);

		g_hash[thread]             = ((uint2*)state)[0];
		g_hash[thread + threads]   = ((uint2*)state)[1];
		g_hash[thread + threads*2] = ((uint2*)state)[2];
		g_hash[thread + threads*3] = ((uint2*)state)[3];
	}
}
#else
#include "cuda_helper.h"
#if __CUDA_ARCH__ < 200
__device__ void* DMatrix;
#endif
__global__ void lyra2v2_gpu_hash_32(const uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
#endif

__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	cuda_get_arch(thr_id);
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];
	uint32_t tpb = TPB52;

	if (cuda_arch[dev_id] > 500) tpb = TPB52;
	else if (cuda_arch[dev_id] == 500) tpb = TPB50;
	else if (cuda_arch[dev_id] >= 350) tpb = TPB35;
	else if (cuda_arch[dev_id] >= 300) tpb = TPB30;
	else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		lyra2v2_gpu_hash_32    <<<grid, block>>> (threads, startNounce, (uint2*)g_hash);
	else
		lyra2v2_gpu_hash_32_v3 <<<grid, block>>> (threads, startNounce, (uint2*)g_hash);

	//MyStreamSynchronize(NULL, order, thr_id);
}
