extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"


static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static uint64_t *d_hash2[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);
extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* matrix);

extern void bmw256_setTarget(const void *ptarget);
extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces);

void lyra2v2_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);

	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2v2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] > 500 && !is_windows()) ? 18 : 17;
	unsigned int defthr = 1U << intensity;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, defthr);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		//hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//if (opt_n_gputhreads == 1)
		//	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		if (device_sm[device_map[thr_id]] < 300) {
			applog(LOG_ERR, "Device SM 3.0 or more recent required!");
			proper_exit(1);
			return -1;
		}

		// DMatrix
		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 16 * 4 * 4 * sizeof(uint64_t) * throughput));
		lyra2v2_cpu_init(thr_id, throughput, d_hash2[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)throughput * 32));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);
	bmw256_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonces[2] = { 0, 0 };

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);

		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonces);

		if (foundNonces[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonces[0]);
			lyra2v2_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was another one...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonces[1] != 0)
				{
					pdata[21] = foundNonces[1];
					res++;
				}
				pdata[19] = foundNonces[0];
				MyStreamSynchronize(NULL, 0, device_map[thr_id]);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && (max_nonce > ((uint64_t)(pdata[19]) + throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	MyStreamSynchronize(NULL, 0, device_map[thr_id]);
	return 0;
}
