#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis, tpruvot)
 */
extern "C" {
#include "sph/sph_whirlpool.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS] = { 0 };

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_cpu_free(int thr_id);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce);
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	// compress the 48 first bytes of the hash to 32
	for (int i = 0; i < 32; i++) {
		hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirlx(int thr_id,  struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int intensity = is_windows() ? 20 : 22;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		whirlpoolx_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, ptarget);
	whirlpoolx_precompute(thr_id);

	do {
		uint32_t foundNonce = whirlpoolx_cpu_hash(thr_id, throughput, pdata[19]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			whirlxHash(vhash64, endiandata);

			*hashes_done = pdata[19] - first_nonce + throughput;

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				work_set_target_ratio(work, vhash64);
				pdata[19] = foundNonce;
				return 1;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		pdata[19] += throughput;

		if (((uint64_t)pdata[19]+throughput) >= max_nonce) {
			break;
		}

	} while (!work_restart[thr_id].restart);

	*(hashes_done) = pdata[19] - first_nonce + 1;

	return 0;
}

// cleanup
extern "C" void free_whirlx(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	whirlpoolx_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
