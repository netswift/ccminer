#include "hip/hip_runtime.h"
/**
 * bmw-256 MDT
 * tpruvot - 2015
 */
extern "C" {
#include "sph/sph_bmw.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void bmw256_midstate_init(int thr_id, uint32_t threads);
extern void bmw256_setBlock_80(int thr_id, void *pdata);
extern void bmw256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int swap);

extern uint32_t cuda_check_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash);

// CPU Hash
extern "C" void bmw_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_bmw256_context ctx;

	sph_bmw256_init(&ctx);
	sph_bmw256(&ctx, input, 80);
	sph_bmw256_close(&ctx, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

extern "C" int scanhash_bmw(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	bool swapnonce = true;
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 21);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0005;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		cuda_check_cpu_init(thr_id, throughput);
		bmw256_midstate_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	bmw256_setBlock_80(thr_id, (void*)endiandata);

	cuda_check_cpu_setTarget(ptarget);

	do {
		bmw256_cpu_hash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id], (int) swapnonce);
		uint32_t foundNonce = cuda_check_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];
			endiandata[19] = swab32_if(foundNonce, swapnonce);
			bmw_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				*hashes_done = foundNonce - first_nonce + 1;
				pdata[19] = swab32_if(foundNonce,!swapnonce);
				return 1;
			}
			else {
				applog(LOG_DEBUG, "GPU #%d: result for nounce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		if ((uint64_t) throughput + pdata[19] > max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
